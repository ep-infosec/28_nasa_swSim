#include "hip/hip_runtime.h"
// Manager Implementation

#include <string>
#include <vector>
#include <unordered_map>
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <iostream>
#include <stdexcept>

#define CUDA_AWARE_MPI TRUE
//
#define _GOCUSPARSE

#ifdef _GOCUSPARSE
    #include <hipsparse.h>
#endif
#include <hip/hip_runtime.h>

#ifdef _CUBLAS
    #include <hipblas.h>
#endif

/*contains: name, Nrows, Ncols, Value array, 
            column index array, row pointer array*/
#include "SparseMatrix_base.hpp"

/*contains: name, value array, "comms info"*/
#include "DenseVector_base.hpp"

/*contains: name, value array, index array*/
#include "SparseVector_base.hpp"

/*contains: helpers regarding error flags, etc*/
#include "KernelComposerHelpers.hpp"

/*contains: self class definition*/
#include "KernelComposerManager.hpp"

# ifndef _CUBLAS
    __global__
    void DAXPY(double* X, double* Y,size_t N,double a){
        int tidx=threadIdx.x+blockDim.x*blockIdx.x;
        if(tidx<N){
            Y[tidx]+=a*X[tidx];
        };
    };
#endif
#ifndef _GOCUSPARSE
    __global__
    void KCSpMV(double alpha,
                swSim::KernelComposerManager::MappedSparseMatrix A,
                swSim::KernelComposerManager::MappedDenseVector X,
                double beta,
                swSim::KernelComposerManager::MappedDenseVector Y){
        int tidx=threadIdx.x+blockDim.x*blockIdx.x;
        int aidx;
        if(tidx<(Y.vector_length-1)){
            Y.vector_d[tidx]=beta*Y.vector_d[tidx];
            for(int midx=A.rowpointers_d[tidx];
                    midx<A.rowpointers_d[tidx+1];midx++){
                aidx=A.columns_d[midx];
                Y.vector_d[tidx]+=alpha*A.values_d[midx]*X.vector_d[aidx];
            };
        }else if(tidx==(Y.vector_length-1)){
            Y.vector_d[tidx]=beta*Y.vector_d[tidx];
            for(int midx=A.rowpointers_d[tidx];
                    midx<A.num_matrix_entries;midx++){
                aidx=A.columns_d[midx];
                Y.vector_d[tidx]+=alpha*A.values_d[midx]*X.vector_d[aidx];
            };    
        };


    };
#endif

// wrapper to print Cuda errors
inline void CHECK_CUDA(hipError_t err){if(hipSuccess!=err){printf("%s\n",hipGetErrorName(err));throw std::runtime_error("Cuda runtime error.");};};

swSim::KernelComposerManager::~KernelComposerManager(){
    if(deviceIsCommitted){
        release();
    };
};

void swSim::KernelComposerManager::release(){
    CHECK_CUDA(hipSetDevice(device));
    if(deviceIsCommitted){
        for(auto it=dense_vectors.begin();
                 it!=dense_vectors.end();it++){
            //release cusparse_vector
            hipsparseDestroyDnVec(it->second.cusparse_vector);
            if(it->second.vector_d){
                CHECK_CUDA(hipFree(it->second.vector_d));
                it->second.vector_d=NULL;
            };
            it->second.vector_length=0;
        };
        for(auto it=sparse_vectors.begin();
                 it!=sparse_vectors.end();it++){
            //release cusparse vector
            hipsparseDestroySpVec(it->second.cusparse_vector);
            if(it->second.vector_d){
                CHECK_CUDA(hipFree(it->second.vector_d));
                it->second.vector_d=NULL;
            };
            if(it->second.idx_d){
                CHECK_CUDA(hipFree(it->second.idx_d));
                it->second.idx_d=NULL;
            };
            it->second.compressed_length=0;
        };
        for(auto it=sparse_matrices.begin();
                 it!=sparse_matrices.end();it++){
            //release cusparse matrix
            hipsparseDestroySpMat(it->second.cusparse_matrix);
            if(it->second.values_d){
                CHECK_CUDA(hipFree(it->second.values_d));
                it->second.values_d=NULL;
            };
            if(it->second.columns_d){
                CHECK_CUDA(hipFree(it->second.columns_d));
                it->second.columns_d=NULL;
            };
            if(it->second.rowpointers_d){
                CHECK_CUDA(hipFree(it->second.rowpointers_d));
                it->second.rowpointers_d=NULL;
            };
            it->second.num_matrix_entries=0;
            it->second.rowpointer_length=0;
        };
        for(auto it=operations.begin();
                 it!=operations.end();it++){
            if(it->buffer_d){
                CHECK_CUDA(hipFree(it->buffer_d));
                it->buffer_d=NULL;
            };
            it->buffer_size=0;
        };
        hipsparseDestroy(sphandle);
        
        #ifdef _CUBLAS
            hipblasDestroy(blashandle);
        #endif
        
        deviceIsCommitted=false;
    };
};
void swSim::KernelComposerManager::commit(int device_id){
    int ndev;
    CHECK_CUDA(hipGetDeviceCount(&ndev));
    if(device_id>=ndev){throw std::runtime_error("device_id exceeds number of GPUs on system");};
    device=device_id;
    commit();
};

void swSim::KernelComposerManager::commit(){
    CHECK_CUDA(hipSetDevice(device));
    if(!deviceIsCommitted){
        hipsparseCreate(&sphandle);

        #ifdef _CUBLAS
            hipblasCreate(&blashandle);
        #endif

        for(auto it=dense_vectors.begin();
                 it!=dense_vectors.end();it++){
            if(it->second.vector_d){
                CHECK_CUDA(hipFree(it->second.vector_d));
                it->second.vector_d=NULL;
            };
            it->second.vector_length=it->second.vector->getArrayLength();
            //hipMalloc(&it->second.vector_d,it->second.vector_length*sizeof(double));
            //hipMemcpy(it->second.vector_d,it->second.vector->getValueArray(),it->second.vector_length*sizeof(double),hipMemcpyHostToDevice);
            CHECK_CUDA(hipMallocManaged(&it->second.vector_d,
                       it->second.vector_length*sizeof(double)));
            memcpy(it->second.vector_d,it->second.vector->getValueArray(),it->second.vector_length*sizeof(double));
            hipsparseCreateDnVec(&(it->second.cusparse_vector),
                                it->second.vector_length,
                                it->second.vector_d,HIP_R_64F);
        };
        for(auto it=sparse_vectors.begin();
                 it!=sparse_vectors.end();it++){
            if(it->second.vector_d){
                CHECK_CUDA(hipFree(it->second.vector_d));
                it->second.vector_d=NULL;
            };
            it->second.compressed_length=it->second.vector->getNonZeroLength();
            //hipMalloc(&it->second.vector_d,it->second.compressed_length*sizeof(double));
            //hipMemcpy(it->second.vector_d,it->second.vector->getValueArray(),it->second.compressed_length*sizeof(double),hipMemcpyHostToDevice);
            CHECK_CUDA(hipMallocManaged(&it->second.vector_d,it->second.compressed_length*sizeof(double)));
            memcpy(it->second.vector_d,it->second.vector->getValueArray(),it->second.compressed_length*sizeof(double));
            if(it->second.idx_d){
                CHECK_CUDA(hipFree(it->second.idx_d));
                it->second.idx_d=NULL;
            };
            //hipMalloc(&it->second.idx_d,it->second.compressed_length*sizeof(int));
            //hipMemcpy(it->second.idx_d,it->second.vector->getIndexArray(),it->second.compressed_length*sizeof(int),hipMemcpyHostToDevice);
            CHECK_CUDA(hipMallocManaged(&it->second.idx_d,it->second.compressed_length*sizeof(int)));
            memcpy(it->second.idx_d,it->second.vector->getIndexArray(),it->second.compressed_length*sizeof(int));
            //build cusparse vector
            hipsparseCreateSpVec(&(it->second.cusparse_vector),
                                it->second.vector->getFullLength(),
                                it->second.compressed_length,
                                it->second.idx_d,it->second.vector_d,
                                HIPSPARSE_INDEX_32I,
                                HIPSPARSE_INDEX_BASE_ZERO,
                                HIP_R_64F);
        };
        for(auto it=sparse_matrices.begin();
                 it!=sparse_matrices.end();it++){
            if(it->second.values_d){
                CHECK_CUDA(hipFree(it->second.values_d));
                it->second.values_d=NULL;
            };
            it->second.num_matrix_entries=
                        it->second.matrix->getNonZeroLength();
            //hipMalloc(&it->second.values_d,it->second.num_matrix_entries*sizeof(double));
            //hipMemcpy(it->second.values_d,it->second.matrix->getValueArray(),it->second.num_matrix_entries*sizeof(double),hipMemcpyHostToDevice);
            CHECK_CUDA(hipMallocManaged(&it->second.values_d,it->second.num_matrix_entries*sizeof(double)));
            memcpy(it->second.values_d,it->second.matrix->getValueArray(),it->second.num_matrix_entries*sizeof(double));
            if(it->second.columns_d){
                CHECK_CUDA(hipFree(it->second.columns_d));
                it->second.columns_d=NULL;
            };
            //hipMalloc(&it->second.columns_d,it->second.num_matrix_entries*sizeof(int));
            //hipMemcpy(it->second.columns_d,it->second.matrix->getColumnIndexArray(),it->second.num_matrix_entries*sizeof(int),hipMemcpyHostToDevice);
            CHECK_CUDA(hipMallocManaged(&it->second.columns_d,it->second.num_matrix_entries*sizeof(int)));
            memcpy(it->second.columns_d,it->second.matrix->getColumnIndexArray(),it->second.num_matrix_entries*sizeof(int));
            if(it->second.rowpointers_d){
                CHECK_CUDA(hipFree(it->second.rowpointers_d));
                it->second.rowpointers_d=NULL;
            };
            it->second.rowpointer_length=
                    it->second.matrix->getRowpointerLength();
            //hipMalloc(&it->second.rowpointers_d,it->second.rowpointer_length*sizeof(int));
            //hipMemcpy(it->second.rowpointers_d,it->second.matrix->getRowpointerArray(),it->second.rowpointer_length*sizeof(int),hipMemcpyHostToDevice);
            CHECK_CUDA(hipMallocManaged(&it->second.rowpointers_d,it->second.rowpointer_length*sizeof(int)));
            memcpy(it->second.rowpointers_d,it->second.matrix->getRowpointerArray(),it->second.rowpointer_length*sizeof(int));
            
            //eigen doesn't include this value at tail-end charlie
            it->second.rowpointers_d[it->second.rowpointer_length-1]=it->second.matrix->getColumnCount();
            
            //build cusparse matrix
            hipsparseCreateCsr(&(it->second.cusparse_matrix),
                        it->second.matrix->getRowCount(),
                        it->second.matrix->getColumnCount(),
                        it->second.num_matrix_entries,
                        it->second.rowpointers_d,
                        it->second.columns_d,it->second.values_d,
                        HIPSPARSE_INDEX_32I, 
                        HIPSPARSE_INDEX_32I,
                        HIPSPARSE_INDEX_BASE_ZERO, 
                        HIP_R_64F);
        };
        for(auto it=operations.begin();
                 it!=operations.end();it++){
            if(it->buffer_d){
                CHECK_CUDA(hipFree(it->buffer_d));
                it->buffer_d=NULL;
                it->buffer_size=0;
            };
            if(it->T==SPMV){
                hipsparseSpMV_bufferSize(sphandle,
                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    &(it->alpha),
                    sparse_matrices.
                        at(it->matrix_name.c_str()).
                        cusparse_matrix,
                    dense_vectors.
                        at(it->argument_name.c_str()).
                        cusparse_vector,
                    &(it->beta),
                    dense_vectors.
                        at(it->result_name.c_str()).
                        cusparse_vector,
                    HIP_R_64F,HIPSPARSE_MV_ALG_DEFAULT,
                    &(it->buffer_size));
                CHECK_CUDA(hipMallocManaged(&(it->buffer_d),(it->buffer_size>0)?(it->buffer_size):1));
            };
        };
        deviceIsCommitted=true;
    };
};
                             
void swSim::KernelComposerManager::execute(int tidx){
    CHECK_CUDA(hipSetDevice(device));
    if(deviceIsCommitted){
        for(auto it=operations.begin();
                 it!=operations.end();it++){
            switch(it->T){
                case(NOOP):
                break;
                case(SPMV):
                #ifdef _GOCUSPARSE
                //call hipsparseSpMV();
                hipsparseSpMV(sphandle,
                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    &(it->alpha),
                    sparse_matrices.
                        at(it->matrix_name).
                        cusparse_matrix,
                    dense_vectors.
                        at(it->argument_name).
                        cusparse_vector,
                    &(it->beta),
                    dense_vectors.
                        at(it->result_name).
                        cusparse_vector,
                    HIP_R_64F,HIPSPARSE_MV_ALG_DEFAULT,
                    &(it->buffer_d));
                #else
                int device;
                hipGetDevice(&device);
                printf("%i: %s=%f*%s*%s+%f*%s\n",device,
                        it->result_name.c_str(),it->alpha,
                        it->matrix_name.c_str(),
                        it->argument_name.c_str(),it->beta,
                        it->result_name.c_str());
KCSpMV<<<dense_vectors.at(it->argument_name).vector_length/32+1,32>>>(it->alpha,sparse_matrices.at(it->matrix_name),
                       dense_vectors.at(it->argument_name),
                       it->beta,dense_vectors.at(it->result_name));
                #endif
                CHECK_CUDA(hipDeviceSynchronize());
                break;
                case(SXPY):
                //call aSpXPY
                if(it->scale->size()>tidx){
                hipsparseDaxpyi(sphandle,
                        sparse_vectors.
                            at(it->argument_name).
                            compressed_length,
                        &(it->scale->at(tidx)),
                        sparse_vectors.
                            at(it->argument_name).
                            vector_d,
                        sparse_vectors.
                            at(it->argument_name).
                            idx_d,
                        dense_vectors.
                            at(it->result_name).
                            vector_d,
                        HIPSPARSE_INDEX_BASE_ZERO);
                };
                CHECK_CUDA(hipDeviceSynchronize());
                break;
                case(DXPY):
                //call AXPY
                #ifdef _CUBLAS
                    hipblasDaxpy(blashandle,
                            dense_vectors.
                                at(it->result_name).
                                vector_length,
                            &(it->alpha),
                            dense_vectors.
                                at(it->argument_name).
                                vector_d,1,
                            dense_vectors.
                                at(it->result_name).
                                vector_d,1);
                #else
DAXPY<<<dense_vectors.at(it->argument_name).vector_length/32+1,32>>>
                    (dense_vectors.at(it->argument_name).vector_d,
                     dense_vectors.at(it->result_name).vector_d,
                     dense_vectors.at(it->result_name).vector_length,
                     (it->alpha));
                #endif
                    CHECK_CUDA(hipDeviceSynchronize());
                break;
                case(UPDT):
                //call from host update device data
                if(sparse_vectors.find(it->argument_name)!=
                    sparse_vectors.end()){
                        //check host with tidx for update readiness
                        sparse_vectors.at(it->argument_name).
                                vector->update(tidx);
                        setSparseVectorData(sparse_vectors.
                                        at(it->argument_name));
                };
                break;
                case(COMM):
                if(dense_vectors.find(it->argument_name)!=
                    dense_vectors.end()){
                        #ifdef CUDA_AWARE_MPI
dense_vectors.at(it->argument_name.c_str()).vector->sendrecv_MPI(
    dense_vectors.at(it->argument_name.c_str()).vector_d);
                        #else
                            dense_vectors.
                                at(it->argument_name.c_str()).
                                    vector->sendrecv_MPI();
                        #endif
                };
                break;
                default:
                //pop error?
                break;
            };
        };
    };
};

void swSim::KernelComposerManager::
            getDenseVectorData(MappedDenseVector& vector){
    CHECK_CUDA(hipSetDevice(device));
    if(deviceIsCommitted){
        if(vector.vector->getArrayLength()==
                    vector.vector_length){
            //hipMemcpy(vector.vector->getValueArray(),vector.vector_d,vector.vector_length*sizeof(double),hipMemcpyDeviceToHost);
            memcpy(vector.vector->getValueArray(),vector.vector_d,vector.vector_length*sizeof(double));
            //for(int k=0;k<vector.vector_length;k++){//printf("%f %f\n",vector.vector->getValueArray()[k],vector.vector_d[k]);};
        };
    };
};
void swSim::KernelComposerManager::
            getSparseVectorData(MappedSparseVector& vector){
    CHECK_CUDA(hipSetDevice(device));
    if(deviceIsCommitted){
        if(vector.vector->getNonZeroLength()==
                    vector.compressed_length){
            //hipMemcpy(vector.vector->getValueArray(),vector.vector_d,vector.compressed_length*sizeof(double),hipMemcpyDeviceToHost);
            memcpy(vector.vector->getValueArray(),vector.vector_d,vector.compressed_length*sizeof(double));
        };
    };
};
void swSim::KernelComposerManager::
            getSparseMatrixData(MappedSparseMatrix& matrix){
    CHECK_CUDA(hipSetDevice(device));
    if(deviceIsCommitted){
        if(matrix.matrix->getNonZeroLength()==
                    matrix.num_matrix_entries){
            //hipMemcpy(matrix.matrix->getValueArray(),matrix.values_d,matrix.num_matrix_entries*sizeof(double));
            memcpy(matrix.matrix->getValueArray(),matrix.values_d,matrix.num_matrix_entries*sizeof(double));
        };
    };
};
void swSim::KernelComposerManager::
            setDenseVectorData(MappedDenseVector& vector){
    CHECK_CUDA(hipSetDevice(device));
    if(deviceIsCommitted){
        if(vector.vector->getArrayLength()==
                    vector.vector_length){
            //hipMemcpy(vector.vector_d,vector.vector->getValueArray(),vector.vector_length*sizeof(double),hipMemcpyHostToDevice);
            memcpy(vector.vector_d,vector.vector->getValueArray(),vector.vector_length*sizeof(double));
        };
    };
};
void swSim::KernelComposerManager::
            setSparseVectorData(MappedSparseVector& vector){
    CHECK_CUDA(hipSetDevice(device));
    if(deviceIsCommitted){
        if(vector.vector->getNonZeroLength()==
                    vector.compressed_length){
            //hipMemcpy(vector.vector_d,vector.vector->getValueArray(),vector.compressed_length*sizeof(double),hipMemcpyHostToDevice);
            memcpy(vector.vector_d,vector.vector->getValueArray(),vector.compressed_length*sizeof(double));
        };
    };
};
void swSim::KernelComposerManager::
            setSparseMatrixData(MappedSparseMatrix& matrix){
    CHECK_CUDA(hipSetDevice(device));
    if(deviceIsCommitted){
        if(matrix.matrix->getNonZeroLength()==
                    matrix.num_matrix_entries){
            //hipMemcpy(matrix.values_d,matrix.matrix->getValueArray(),matrix.num_matrix_entries*sizeof(double),hipMemcpyHostToDevice);
            memcpy(matrix.values_d,matrix.matrix->getValueArray(),matrix.num_matrix_entries*sizeof(double));
        };
    };
};
